#include "hip/hip_runtime.h"
#include "Bool.h"
#include "UserSettings.h"
#include "MathLinearStructures.cuh"
#include "MathLinear.cuh"
#include <malloc.h>
#include <assert.h>
#include <math.h>

#ifdef _WIN32
#include <Windows.h>
#endif

//temporary for printf
#include <stdio.h>

#include ""
#ifndef _CUDA_RUNTIME_
#define _CUDA_RUNTIME_
#include "hip/hip_runtime.h"
#endif

#include "math.h"

//#define PRECISION 0.00001
#define PRECISION 0.1


#define SGN(a)		( (a)>0 ? 1 : ((a)<0?-1:0) )

#define Z_BETWEEN_X_AND_Y(z,x,y)	((x)>(y) ? ((z)>=(y))&&((z)<=(x)) : ((z)<=(y))&&((z)>=(x)))
//bugfix
#define X_NEAR_Y(x,y)	Z_BETWEEN_X_AND_Y((x),(y)-PRECISION,(y)+PRECISION)

#define NEAR_0(x)	X_NEAR_Y((x),0)

#define POINT_IN_PLANE(point, plane)	\
			(NEAR_0(  (  (plane)[0]*(point)[0] + (plane)[1]*(point)[1] + (plane)[2]*(point)[2] + (plane)[3]  )  ))

//is a helper-macros for ON_ONE_SIDE
#define LINE(x, y, p1x_line, p1y_line, p2x_line, p2y_line)	\
			(   (x)*((p1y_line)-(p2y_line))  -  (y)*((p1x_line)-(p2x_line))  +  (p1x_line)*(p2y_line)-(p2x_line)*(p1y_line)  )

//function for four points to define one what side is situaded first and second point relarive to line (third and fourth points)
#define ON_ONE_SIDE(p1x, p1y, p2x, p2y, p1x_line, p1y_line, p2x_line, p2y_line)		\
			((  SGN( LINE(p1x, p1y, p1x_line, p1y_line, p2x_line, p2y_line) )*			\
			    SGN( LINE(p2x, p2y, p1x_line, p1y_line, p2x_line, p2y_line) )   )>=0)

//define plane for projection, coordinate's - out parametrs
#define CHOISE_PROJECTION(polygon_plane, coordinate1, coordinate2)	\
			if ((polygon_plane)[2])  {  (coordinate1)=0;  (coordinate2)=1;  }					\
			else if((polygon_plane)[1]) { (coordinate1)=0;  (coordinate2)=2; }				\
			else {  (coordinate1)=1;  (coordinate2)=2;  }

//is point in segment? for 3D ok; points - are pointers
#define POINT_IN_SEGMENT_3D(point, seg_point1, seg_point2)	\
			(  ((seg_point1)[0]!=(seg_point2)[0])  ?  ( Z_BETWEEN_X_AND_Y((point)[0],(seg_point1)[0],(seg_point2)[0]) )				\
			:  ((seg_point1)[1]!=(seg_point2)[1]   ?  ( Z_BETWEEN_X_AND_Y((point)[1],(seg_point1)[1],(seg_point2)[1]) )				\
												   :  ( Z_BETWEEN_X_AND_Y((point)[2],(seg_point1)[2],(seg_point2)[2]) ) ) )

#define POINT_IN_SEGMENT_2D(px,py, sp1x,sp1y, sp2x,sp2y)	\
			( ((sp1x)!=(sp2x))  ?  ( Z_BETWEEN_X_AND_Y((px),(sp1x),(sp2x)) )					\
								:  ( Z_BETWEEN_X_AND_Y((py),(sp1y),(sp2y)) ))

//is segments are intersected (segments on one line)? for 1D and 2D and 3D ok
#define SEGMENTS_INTERSECT_ON_LINE_3D(seg1p1, seg1p2, seg2p1, seg2p2)	\
			(  POINT_IN_SEGMENT_3D((seg1p1),(seg2p1),(seg2p2))									\
			|| POINT_IN_SEGMENT_3D((seg1p2),(seg2p1),(seg2p2))									\
			|| POINT_IN_SEGMENT_3D((seg2p1),(seg1p1),(seg1p2))									\
			|| POINT_IN_SEGMENT_3D((seg2p2),(seg1p1),(seg1p2)) )

//define line coefficients abc on the plane through two points: p1 and p2; a,b,c - out parametrs must be declared before
#define LINE_DEFINE(a, b, c, p1x, p1y, p2x, p2y)	{  a=p1y-p2y;   b=p2x-p1x;   c=-a*p1x-b*p1y;  }

//divide segment into parts in the ratio (lyambda):(1-lyambda) from the point1 to point2
//return x0, y0, z0
#define SEGMENT_DIVIDE(x0, y0, z0,  lyambda,  x1, y1, z1,   x2, y2, z2)							\
								{ (x0) = (x1) + (lyambda)*((x2)-(x1));									\
								  (y0) = (y1) + (lyambda)*((y2)-(y1));									\
								  (z0) = (z1) + (lyambda)*((z2)-(z1));}

//get the centroid coordinates in the triangle p1, p2, p3
//p1, p2, p3 should situated in one plane!
//return out: x0, y0, z0
#define GET_CENTROID(x0,y0,z0,  x1,y1,z1,  x2,y2,z2,  x3,y3,z3)			SEGMENT_DIVIDE((x0),(y0),(z0),  2.0/3.0,    (x1), (y1), (z1),   \
																				((x2)+(x3))/2.0, ((y2)+(y3))/2.0, ((z2)+(z3))/2.0   )

//get the centroid coordinates in the quadropolygon p1, p2, p3, p4
//p1, p2, p3, p4 should be in one plane!
//quadropolygon should be convex!
//return out: x0, y0, z0
#define GET_CENTROID_4(x0,y0,z0, x1,y1,z1, x2,y2,z2, x3,y3,z3, x4,y4,z4)								\
								{ (x0) = ((x1)+(x2)+(x3)+(x4))/4.0;					\
								  (y0) = ((y1)+(y2)+(y3)+(y4))/4.0;					\
								  (z0) = ((z1)+(z2)+(z3)+(z4))/4.0;}


//if we have segment [(1,1,1);(2,2,2)] => [(1.0003,1.0003,1.0003);(2,2,2)] whereif eps = 0.0001 (show PRECISION); on 3*eps value change
//change only first point (!) => input: x1,y1,z1 ,x2,y2,z2; output: x1,y1,z1 (just change x1,y1,z1)
#define DECREASE_SEGMENT_TO_INTERVAL_3D(x1,y1,z1,  x2,y2,z2)									\
								{ (x1) = (x1) + (3.0*(PRECISION)*( (x2) - (x1) )/ ( ((x2)-(x1))*((x2)-(x1)) + ((y2)-(y1))*((y2)-(y1)) + ((z2)-(z1))*((z2)-(z1)) ));			\
								  (y1) = (y1) + (3.0*(PRECISION)*( (y2) - (y1) )/ ( ((x2)-(x1))*((x2)-(x1)) + ((y2)-(y1))*((y2)-(y1)) + ((z2)-(z1))*((z2)-(z1)) ));			\
								  (z1) = (z1) + (3.0*(PRECISION)*( (z2) - (z1) )/ ( ((x2)-(x1))*((x2)-(x1)) + ((y2)-(y1))*((y2)-(y1)) + ((z2)-(z1))*((z2)-(z1)) ));}

//interval ro between A and B
#define INTERVAL_BETWEEN_A_AND_B(ax,ay,az, bx,by,bz)										\
								 sqrt(   ((ax)-(bx))*((ax)-(bx)) +  ((ay)-(by))*((ay)-(by)) +  ((az)-(bz))*((az)-(bz))  )


//return true if segments intersected
__host__ __device__
bool IsSegmentsIntersected2D(real seg1p1x, real seg1p1y, real seg1p2x, real seg1p2y,
							 real seg2p1x, real seg2p1y, real seg2p2x, real seg2p2y)
{
	real a1,b1,c1, a2,b2,c2, kramer_determinant, IntersectionPoint[2];
	LINE_DEFINE(a1,b1,c1, seg1p1x,seg1p1y, seg1p2x,seg1p2y);
	LINE_DEFINE(a2,b2,c2, seg2p1x,seg2p1y, seg2p2x,seg2p2y);
	kramer_determinant=a1*b2-b1*a2;
	if (kramer_determinant) { //not parallel and not union
		IntersectionPoint[0]=(b1*c2-c1*b2)/kramer_determinant;
		IntersectionPoint[1]=(c1*a2-a1*c2)/kramer_determinant;
		if (POINT_IN_SEGMENT_2D(IntersectionPoint[0],IntersectionPoint[1], seg1p1x,seg1p1y, seg1p2x, seg1p2y))
			if (POINT_IN_SEGMENT_2D(IntersectionPoint[0],IntersectionPoint[1], seg2p1x,seg2p1y, seg2p2x, seg2p2y))
				return true;
	}
	else if (!(a1!=0 ? a1*c2-c1*a2: b1*c2-c1*b2)) { //equal lines, but what about segments?
		if (POINT_IN_SEGMENT_2D(seg1p1x,seg1p1y,seg2p1x,seg2p1y,seg2p2x,seg2p2y)||
			POINT_IN_SEGMENT_2D(seg1p2x,seg1p2y,seg2p1x,seg2p1y,seg2p2x,seg2p2y)||
			POINT_IN_SEGMENT_2D(seg2p1x,seg2p1y,seg1p1x,seg1p1y,seg1p2x,seg1p2y)||
			POINT_IN_SEGMENT_2D(seg2p2x,seg2p2y,seg1p1x,seg1p1y,seg1p2x,seg1p2y))
			return true;
	}
	return false;
}

//founding plane by array of points
//plane - out parametr; size(plane[])=4
__host__ __device__
void PlaneDefine(CalcFace *face)
{
	___PlaneFrom(face)[0] = ___YPointFrom(face, 1) * ( ___ZPointFrom(face, 2) - ___ZPointFrom(face, 0) ) +
						    ___YPointFrom(face, 0) * ( ___ZPointFrom(face, 1) - ___ZPointFrom(face, 2) ) +
					   	    ___YPointFrom(face, 2) * ( ___ZPointFrom(face, 0) - ___ZPointFrom(face, 1) );

	___PlaneFrom(face)[1] = - (
							  ___XPointFrom(face, 1) * ( ___ZPointFrom(face, 2) - ___ZPointFrom(face, 0) ) +
							  ___XPointFrom(face, 0) * ( ___ZPointFrom(face, 1) - ___ZPointFrom(face, 2) ) +
							  ___XPointFrom(face, 2) * ( ___ZPointFrom(face, 0) - ___ZPointFrom(face, 1) ) );

	___PlaneFrom(face)[2] =   ___XPointFrom(face, 1) * ( ___YPointFrom(face, 2) - ___YPointFrom(face, 0) ) +
							  ___XPointFrom(face, 0) * ( ___YPointFrom(face, 1) - ___YPointFrom(face, 2) ) +
							  ___XPointFrom(face, 2) * ( ___YPointFrom(face, 0) - ___YPointFrom(face, 1) );

	___PlaneFrom(face)[3] = - ___XPointFrom(face, 0) * ___PlaneFrom(face)[0]
							- ___YPointFrom(face, 0) * ___PlaneFrom(face)[1]
							- ___ZPointFrom(face, 0) * ___PlaneFrom(face)[2] ;
}

//only for convex polygons!
//need non-convex -to-> convex use operator before
__host__ __device__
void DefineConvexSquare(CalcFace *face)
{
	real a, b, c, p;
	integer i;
	face->Square = 0.0;
	for (i=2; i < ___DimOf(face); i++)
	{
		//a = ro between 0 and i-1
		a = INTERVAL_BETWEEN_A_AND_B(___XPointFrom(face, 0), ___YPointFrom(face, 0), ___ZPointFrom(face, 0), 		___XPointFrom(face, i-1), ___YPointFrom(face, i-1), ___ZPointFrom(face, i-1));
		//b = ro between 0 and i-1
		b = INTERVAL_BETWEEN_A_AND_B(___XPointFrom(face, 0), ___YPointFrom(face, 0), ___ZPointFrom(face, 0),            ___XPointFrom(face, i), ___YPointFrom(face, i), ___ZPointFrom(face, i));
		//c = ro between i and i-1
		c = INTERVAL_BETWEEN_A_AND_B(___XPointFrom(face, i), ___YPointFrom(face, i), ___ZPointFrom(face, i),            ___XPointFrom(face, i-1), ___YPointFrom(face, i-1), ___ZPointFrom(face, i-1));
		p = (a+b+c) /2;
		face->Square += sqrt(p*(p-a)*(p-b)*(p-c));
	}
}

//true if Quadrilateral without intersection
//only for 4-points Quadrilateral
//TODO:need for all non-convex polyogns: non-convex -to-> convex.
__host__ __device__
void CorrectQuadrilateralSimplisity(CalcFace *face)
{
	//we need in proection:
	unsigned char coordinate1, coordinate2;
	bool flag;
	CHOISE_PROJECTION(___PlaneFrom(face), coordinate1, coordinate2);
	if (IsSegmentsIntersected2D(___XYZPointFrom(face, 0, coordinate1), ___XYZPointFrom(face, 0, coordinate2),
								___XYZPointFrom(face, 1, coordinate1), ___XYZPointFrom(face, 1, coordinate2),
								___XYZPointFrom(face, 2, coordinate1), ___XYZPointFrom(face, 2, coordinate2),
								___XYZPointFrom(face, 3, coordinate1), ___XYZPointFrom(face, 3, coordinate2)))
			flag=false;
	else if (IsSegmentsIntersected2D(___XYZPointFrom(face, 0, coordinate1), ___XYZPointFrom(face, 0, coordinate2),
									 ___XYZPointFrom(face, 3, coordinate1), ___XYZPointFrom(face, 3, coordinate2),
									 ___XYZPointFrom(face, 1, coordinate1), ___XYZPointFrom(face, 1, coordinate2),
									 ___XYZPointFrom(face, 2, coordinate1), ___XYZPointFrom(face, 2, coordinate2)))
			flag=true;
	else return;
	___ExchangeVertices(flag?2:0, 3, face);
}

//size >=3 - is dimension (for 3-polygon or 4-polygon for example)
//you can use it only for 3-dim or 4-dim or any-dim convex
__host__ __device__
Bool PointInPolygon(CalcVertex *point, CalcFace *face)
{
	if (POINT_IN_PLANE((real*)point, ___PlaneFrom(face)))		//\EF\EE\E4 \E8\ED\F2\E5\F0\F4\E5\E9\F1
	{
		//checking point of intersection in polygon (by projection)
		//choose the plane for projection:
		unsigned char coordinate1, coordinate2;
		CHOISE_PROJECTION(___PlaneFrom(face), coordinate1, coordinate2);
		if (___DimOf(face)==4) { 
			//if non-convex 4-dimension
			//need in manual control do different convex and non-convex => copy of function IsSegmentIntersected2D:
			real a1,b1,c1, a2,b2,c2, kramer_determinant, IntersectionPoint[2], triag[2][9]; //\EF\F0\E5\EE\E1\F0\E0\E7\EE\E2\E0\F2\FC \E2 \F2\EE\F7\EA\E8
			bool diag[2];
			LINE_DEFINE(a1,b1,c1,
				___XYZPointFrom(face, 0, coordinate1), ___XYZPointFrom(face, 0, coordinate2),
				___XYZPointFrom(face, 2, coordinate1), ___XYZPointFrom(face, 2, coordinate2));
			LINE_DEFINE(a2,b2,c2,
				___XYZPointFrom(face, 1, coordinate1), ___XYZPointFrom(face, 1, coordinate2),
				___XYZPointFrom(face, 3, coordinate1), ___XYZPointFrom(face, 3, coordinate2));
			kramer_determinant=a1*b2-b1*a2; //diagonal lines intersected in all ways of life
			IntersectionPoint[0]=(b1*c2-c1*b2)/kramer_determinant;
			IntersectionPoint[1]=(c1*a2-a1*c2)/kramer_determinant;
			diag[0]=POINT_IN_SEGMENT_2D(IntersectionPoint[0],IntersectionPoint[1],
				___XYZPointFrom(face, 0, coordinate1), ___XYZPointFrom(face, 0, coordinate2),
				___XYZPointFrom(face, 2, coordinate1), ___XYZPointFrom(face, 2, coordinate2));
			diag[1]=POINT_IN_SEGMENT_2D(IntersectionPoint[0],IntersectionPoint[1],
				___XYZPointFrom(face, 1, coordinate1), ___XYZPointFrom(face, 1, coordinate2),
				___XYZPointFrom(face, 3, coordinate1), ___XYZPointFrom(face, 3, coordinate2));
			//condition of non-convex 4-dimension polygon:
			if (diag[0] != diag[1]) {
				if (!diag[0]) { //it's ok that coordinate3 not used because in recurse function other CHOISE_PROJECTION and coordinate3 never used
					triag[1][0*3+coordinate1] = triag[0][0*3+coordinate1] = ___XYZPointFrom(face, 0, coordinate1);
					triag[1][0*3+coordinate2] = triag[0][0*3+coordinate2] = ___XYZPointFrom(face, 0, coordinate2);
					triag[1][1*3+coordinate1] = triag[0][1*3+coordinate1] = ___XYZPointFrom(face, 2, coordinate1);
					triag[1][1*3+coordinate2] = triag[0][1*3+coordinate2] = ___XYZPointFrom(face, 2, coordinate2);
					triag[0][2*3+coordinate1]= ___XYZPointFrom(face, 1, coordinate1);
					triag[0][2*3+coordinate2]= ___XYZPointFrom(face, 1, coordinate2);
					triag[1][2*3+coordinate1]= ___XYZPointFrom(face, 3, coordinate1);
					triag[1][2*3+coordinate2]= ___XYZPointFrom(face, 3, coordinate2);
				}
				else {
					triag[1][0*3+coordinate1] = triag[0][0*3+coordinate1] = ___XYZPointFrom(face, 1, coordinate1);
					triag[1][0*3+coordinate2] = triag[0][0*3+coordinate2] = ___XYZPointFrom(face, 1, coordinate2);
					triag[1][1*3+coordinate1] = triag[0][1*3+coordinate1] = ___XYZPointFrom(face, 3, coordinate1);
					triag[1][1*3+coordinate2] = triag[0][1*3+coordinate2] = ___XYZPointFrom(face, 3, coordinate2);
					triag[0][2*3+coordinate1]=___XYZPointFrom(face, 2, coordinate1);
					triag[0][2*3+coordinate2]=___XYZPointFrom(face, 2, coordinate2);
					triag[1][2*3+coordinate1]=___XYZPointFrom(face, 0, coordinate1);
					triag[1][2*3+coordinate2]=___XYZPointFrom(face, 0, coordinate2);
				}
				unsigned char k;
				for(unsigned char j=0; j<2; j++) {
					k=0; diag[j]=true;
					while (diag[j]&&(k<3)) {
						if ( !ON_ONE_SIDE(
								___XYZOf(point, coordinate1), ___XYZOf(point, coordinate2),
								triag[j][k*3+coordinate1],triag[j][k*3+coordinate2],
								triag[j][(k+1)%3*3+coordinate1],triag[j][(k+1)%3*3+coordinate2],
								triag[j][(k+2)%3*3+coordinate1],triag[j][(k+2)%3*3+coordinate2]))
							diag[j]=false; //flag==true if point in triangle and false if no
						k++;
					}
				}
				return (diag[0]||diag[1]);	//no recusre in CUDA - no "return (PointInPolygon(point, triag1, polygon_plane, 3)||PointInPolygon(point, triag2, polygon_plane, 3)); //trianglyed it!"
			}
		}
		//DEBUG
		//coordinate1 = 1;
		//coordinate2 = 2;
		for (unsigned char i=0; i < ___DimOf(face); i++) {			
			if ( !ON_ONE_SIDE(
					___XYZOf(point, coordinate1), ___XYZOf(point, coordinate2),
					___XYZPointFrom(face, i, coordinate1),
					___XYZPointFrom(face, i, coordinate2),
					___XYZPointFrom(face, (i+1) % ___DimOf(face), coordinate1),
					___XYZPointFrom(face, (i+1) % ___DimOf(face), coordinate2),
					___XYZPointFrom(face, (i+2) % ___DimOf(face), coordinate1),
					___XYZPointFrom(face, (i+2) % ___DimOf(face), coordinate2))
			   ) return false;
//DEBUG			
			printf("%d for (%f,%f) and (%f,%f)\n", 			
			!ON_ONE_SIDE(
                                        ___XYZOf(point, coordinate1), ___XYZOf(point, coordinate2),
                                        ___XYZPointFrom(face, 0, coordinate1),
                                        ___XYZPointFrom(face, 0, coordinate2),
                                        ___XYZPointFrom(face, (0+1) % ___DimOf(face), coordinate1),
                                        ___XYZPointFrom(face, (0+1) % ___DimOf(face), coordinate2),
                                        ___XYZPointFrom(face, (0+2) % ___DimOf(face), coordinate1),
                                        ___XYZPointFrom(face, (0+2) % ___DimOf(face), coordinate2)),
			___XYZPointFrom(face, 0, coordinate1),  ___XYZPointFrom(face, 0, coordinate2),
			___XYZOf(point, coordinate1), ___XYZOf(point, coordinate2)			
			);
			

		}
		return true;
	}
	return false;
}

//if (segment[point1, point2] interset Polygon)
__host__ __device__
Bool IsSegmentIntersectPolygon(CalcVertex *point1, CalcVertex *point2,
							   CalcFace *face)
{
	real  temp = - ( ___PlaneFrom(face)[0] * ( ___XOf(point2) - ___XOf(point1) ) +
					 ___PlaneFrom(face)[1] * ( ___YOf(point2) - ___YOf(point1) ) +
					 ___PlaneFrom(face)[2] * ( ___ZOf(point2) - ___ZOf(point1) ) );
	CalcVertex IntersectionPoint;
	if (!NEAR_0(temp)) //if polygon and segment are not parallel
	{
			IntersectionPoint.x=(  ___PlaneFrom(face)[3] * (___XOf(point2) - ___XOf(point1)) +
								   ___PlaneFrom(face)[1] * (___YOf(point1)*___XOf(point2) - ___XOf(point1)*___YOf(point2)) +
								   ___PlaneFrom(face)[2] * (___ZOf(point1)*___XOf(point2) - ___XOf(point1)*___ZOf(point2)) ) / temp;
			IntersectionPoint.y=(  ___PlaneFrom(face)[3] * (___YOf(point2) - ___YOf(point1)) +
				                   ___PlaneFrom(face)[0] * (___XOf(point1)*___YOf(point2) - ___YOf(point1)*___XOf(point2)) +
								   ___PlaneFrom(face)[2] * (___ZOf(point1)*___YOf(point2) - ___YOf(point1)*___ZOf(point2)) ) / temp;
			IntersectionPoint.z=(  ___PlaneFrom(face)[3] * (___ZOf(point2) - ___ZOf(point1)) +
				                   ___PlaneFrom(face)[0] * (___XOf(point1)*___ZOf(point2) - ___ZOf(point1)*___XOf(point2)) +
								   ___PlaneFrom(face)[1] * (___YOf(point1)*___ZOf(point2) - ___ZOf(point1)*___YOf(point2)) ) / temp;
			//if IntersectionPoint in segment:
			if (POINT_IN_SEGMENT_3D( (real*)&IntersectionPoint, (real*)point1, (real*)point2 ))
				if (PointInPolygon(&IntersectionPoint, face)) 
							return true;
	}
	else {
		if ((point1->x == point2->x)&&(point1->y == point2->y)&&(point1->z == point2->z))
			return PointInPolygon(point1, face);
		else if (POINT_IN_PLANE((real*)&point1, ___PlaneFrom(face)))
		{//we need in proection:
			unsigned char coordinate1, coordinate2;
			CHOISE_PROJECTION(___PlaneFrom(face), coordinate1, coordinate2);
			for (unsigned char i=0; i < ___DimOf(face); i++)
				if (IsSegmentsIntersected2D( ___XYZOf(point1, coordinate1), ___XYZOf(point1, coordinate2),
						___XYZOf(point2, coordinate1), ___XYZOf(point2, coordinate2),
						___XYZPointFrom(face, i, coordinate1),
						___XYZPointFrom(face, i, coordinate2),
						___XYZPointFrom(face, (i+1)%___DimOf(face), coordinate1),
						___XYZPointFrom(face, (i+1)%___DimOf(face), coordinate2)))
					return true;
		}
	}
	return false;
}


//if (segment[point1, point2] interset Model)
//return number of intersection points
__host__ __device__
integer IsSegmentIntersectModel(CalcVertex *point1, CalcVertex *point2, CalcMesh *mesh)
{
	integer intersections = 0;
	//DEBUG if 	
//	if ((point2->x == -34.475559)&&(point2->y==143.815369)&&(point2->z==-271.077057))
//	{
//		if (IsSegmentIntersectPolygon(point1, point2, &mesh->Faces[23127]))
//		intersections = 101;
//		else intersections = 100;	
//	}
//	else
	for (integer i=0; i < mesh->NumberOfFaces; i++)
	{
		if(IsSegmentIntersectPolygon(point1, point2, &mesh->Faces[i]))
		{
			intersections++;	
		}
	}
	
	return intersections;
}


//TODO PlaneDefine and SquareDefine - it is a calculate procedure. It shoud be on the GPU or CPU differently from CreateCalcMesh
CalcMesh* CreateCalcMesh(ObjMesh *objMesh)
{
	CalcMesh* calcMesh = NULL;
	if (objMesh != NULL)
	{
		integer i,j;
		calcMesh = (CalcMesh*) malloc(sizeof(CalcMesh));
		assert(calcMesh);
		calcMesh->NumberOfFaces = objMesh->m_iNumberOfFaces;
		calcMesh->NumberOfVertices = objMesh->m_iNumberOfVertices;
		calcMesh->TypesOfFaces = (unsigned char*) calloc(calcMesh->NumberOfFaces, sizeof(unsigned char));
		assert(calcMesh->TypesOfFaces);
		calcMesh->VertexArray = (CalcVertex*) calloc(calcMesh->NumberOfVertices, sizeof(CalcVertex));
		assert(calcMesh->VertexArray);
		
		calcMesh->NumberSphereDetalisation = objMesh->m_iNumberSphereDetalisation;
		calcMesh->SpherePolygonRadiosity = (real*) calloc(calcMesh->NumberSphereDetalisation*(calcMesh->NumberSphereDetalisation-1), sizeof(real));
		assert(calcMesh->SpherePolygonRadiosity);
		for (i=0; i < calcMesh->NumberSphereDetalisation*(calcMesh->NumberSphereDetalisation-1); i++)
		{
			calcMesh->SpherePolygonRadiosity[i] = objMesh->m_aSpherePolygonRadiosity[i];	
		}
		
		calcMesh->Lights = NULL;
		for (i=0; i < calcMesh->NumberOfVertices; i++)
		{
			calcMesh->VertexArray[i].x = objMesh->m_aVertexArray[i].x;
			calcMesh->VertexArray[i].y = objMesh->m_aVertexArray[i].y;
			calcMesh->VertexArray[i].z = objMesh->m_aVertexArray[i].z;
		}
		calcMesh->Faces = (CalcFace*) calloc(calcMesh->NumberOfFaces, sizeof(CalcFace));
		//real p,a,b,c;
		for (i=0; i < calcMesh->NumberOfFaces; i++)
		{
			calcMesh->TypesOfFaces[i] = UNDEFINED_VISION;
			calcMesh->Faces[i].VertexCount = objMesh->m_aFaces[i].m_iVertexCount;
			calcMesh->Faces[i].VertexArray = calcMesh->VertexArray;
			for (j=0; j < calcMesh->Faces[i].VertexCount; j++)
			{
				calcMesh->Faces[i].VertexIndices[j] = objMesh->m_aFaces[i].m_aVertexIndices[j];
			}
			
			PlaneDefine(calcMesh->Faces + i);
			//only for convex polygons
			DefineConvexSquare(calcMesh->Faces+i);
		}
		if (objMesh->m_aLights) //only one light in this implementation
		{
			calcMesh->Lights = (CalcVertex*) malloc(sizeof(CalcVertex));
			calcMesh->Lights[0].x = objMesh->m_aLights[0].x;
			calcMesh->Lights[0].y = objMesh->m_aLights[0].y;
			calcMesh->Lights[0].z = objMesh->m_aLights[0].z;
		}
	}
	return calcMesh;
}

void CopyResults(CalcMesh *calcMesh, ObjMesh *objMesh)
{
	//if (objMesh->m_aTypesOfFaces != NULL)
	{
		integer i;
		for (i=0; i < calcMesh->NumberOfFaces; i++)
		{
			objMesh->m_aTypesOfFaces[i] = calcMesh->TypesOfFaces[i];
		}
		for (i=0; i < calcMesh->NumberSphereDetalisation * (calcMesh->NumberSphereDetalisation-1); i++)
		{
			objMesh->m_aSpherePolygonRadiosity[i] = calcMesh->SpherePolygonRadiosity[i];
		}
	}
}


void DeleteCalcMesh(CalcMesh *calcMesh)
{
	if (calcMesh != NULL)
	{
		int i;
		free(calcMesh->Faces);
		free(calcMesh->VertexArray);
		free(calcMesh->TypesOfFaces);
		if (calcMesh->Lights) free(calcMesh->Lights);
		if (calcMesh->SpherePolygonRadiosity) free(calcMesh->SpherePolygonRadiosity);
		free(calcMesh);
		calcMesh = NULL;
	}
}



//to count first faces from light-point (by segments [light, centroid of polygon])
__host__ __device__
void ToCountFirstFaces(CalcVertex *light, CalcMesh *mesh)
{
	CalcVertex centroid;
	//int temp;
	for(integer i = 0; i < mesh->NumberOfFaces; i++ )
	{
		//for triangles-polygons
		if (mesh->Faces[i].VertexCount == 3)
		{



			GET_CENTROID(centroid.x, centroid.y, centroid.z,
				___XPointFrom(mesh->Faces + i, 0), ___YPointFrom(mesh->Faces + i, 0), ___ZPointFrom(mesh->Faces + i, 0),
				___XPointFrom(mesh->Faces + i, 1), ___YPointFrom(mesh->Faces + i, 1), ___ZPointFrom(mesh->Faces + i, 1),
				___XPointFrom(mesh->Faces + i, 2), ___YPointFrom(mesh->Faces + i, 2), ___ZPointFrom(mesh->Faces + i, 2));
			//the face include centroid => intersections >= 1

			//mesh->TypesOfFaces[i]  = IsSegmentIntersectModel(light, &centroid, mesh
			//temp = IsSegmentIntersectModel(light, &centroid, mesh);
			//if (temp > 10) printf("face %d - intersections %d\n", i, temp);

			DECREASE_SEGMENT_TO_INTERVAL_3D(centroid.x, centroid.y, centroid.z,
				light->x, light->y, light->z);

			if (IsSegmentIntersectModel(light, &centroid, mesh) == 0)
				mesh->TypesOfFaces[i] = FIRST_VISION;
			//else if (IsSegmentIntersectModel(light, &centroid, mesh) == 0)
			//	printf("WTF\n");
			else mesh->TypesOfFaces[i] = OTHER_VISION;
		}
		//else;// if (mesh->Faces[i].VertexCount != 3)
			//printf("ERROR! MORE WHEN 3\n");
    }
}





//__global__
//void cudaIsSegmentIntersectModel(CalcVertex *point1, CalcVertex *point2, CalcMesh *mesh, integer numberOfFace)
//{
//	integer intersections = 0;
//	integer j = threadIdx.x + blockIdx.x * blockDim.x;
//	max j should be more then mesh->NumberOfFaces
//	if (j < mesh->NumberOfFaces)
//	{
//		if(IsSegmentIntersectPolygon(point1, point2, &mesh->Faces[j]))
//		{
//			atomicAdd(&intersections, 1);
//		}
//	}
//
//
//	if (intersections == 0)
//		mesh->TypesOfFaces[numberOfFace] = FIRST_VISION;
//	else mesh->TypesOfFaces[numberOfFace] = OTHER_VISION;
//}


__global__
void cudaToCountFirstFaces(CalcVertex light, CalcMesh* mesh, float* ret)
{
	CalcVertex centroid;
	integer i; //intersections, i, j;
	i = threadIdx.x + blockIdx.x * blockDim.x;// + beginFrom;
	//int temp;
	//mesh->NumberOfFaces=1;
	//for(i=0; i< mesh->NumberOfFaces; i++)
	//if ((i >= beginFrom) && (i < beginFrom + 1))
	if(i < mesh->NumberOfFaces)
	{
		//for triangles-polygons
		if (mesh->Faces[i].VertexCount == 3)
		{

			GET_CENTROID(centroid.x, centroid.y, centroid.z,
				___XPointFrom(mesh->Faces + i, 0), ___YPointFrom(mesh->Faces + i, 0), ___ZPointFrom(mesh->Faces + i, 0),
				___XPointFrom(mesh->Faces + i, 1), ___YPointFrom(mesh->Faces + i, 1), ___ZPointFrom(mesh->Faces + i, 1),
				___XPointFrom(mesh->Faces + i, 2), ___YPointFrom(mesh->Faces + i, 2), ___ZPointFrom(mesh->Faces + i, 2));
			//the face include centroid => intersections >= 1

			//change centroid on delta:
			DECREASE_SEGMENT_TO_INTERVAL_3D(centroid.x, centroid.y, centroid.z,
				light.x, light.y, light.z);

			if (IsSegmentIntersectModel(&light, &centroid, mesh) == 0)
				mesh->TypesOfFaces[i] = FIRST_VISION;
			else mesh->TypesOfFaces[i] = OTHER_VISION;
			
				

//2150 for eleham.obj		
//	if (i==2150) 	{
//		*ret = IsSegmentIntersectModel(&light, &centroid, mesh);
		//*ret = light.z;
//			}

		}
    }
if (i==0) *ret = 99;
//__syncthreads();
}


//to count second faces ONLY AFTER first faces has been defined!
//(by segments [centroid of first face polygon; centroid of not first face polygon])
__host__ __device__
	//	for (integer j=0; j < mesh->NumberOfFaces/(65535*prop.maxThreadsPerBlock)+1; j++) 
	void ToCountSecondAndDoubleFaces(CalcMesh *mesh)
	{
		//just only for triangle-polygons now
		integer i,j;
		CalcVertex centroidFirst, centroidOther, temp;
		//for {all first_vision face polygons}
		for (i=0; i < mesh->NumberOfFaces; i++)
			if ((mesh->TypesOfFaces[i] == FIRST_VISION)&&(mesh->Faces[i].VertexCount == 3))
		{
			GET_CENTROID(centroidFirst.x, centroidFirst.y, centroidFirst.z,
				___XPointFrom(mesh->Faces + i, 0), ___YPointFrom(mesh->Faces + i, 0), ___ZPointFrom(mesh->Faces + i, 0),
				___XPointFrom(mesh->Faces + i, 1), ___YPointFrom(mesh->Faces + i, 1), ___ZPointFrom(mesh->Faces + i, 1),
				___XPointFrom(mesh->Faces + i, 2), ___YPointFrom(mesh->Faces + i, 2), ___ZPointFrom(mesh->Faces + i, 2));

			//fo {all other_vision face polygons}
			for (j=0; j < mesh->NumberOfFaces; j++)
				if ( ( (mesh->TypesOfFaces[j] == OTHER_VISION) || (mesh->TypesOfFaces[j] == FIRST_VISION) )
					&&(mesh->Faces[j].VertexCount == 3))
					{
						GET_CENTROID(centroidOther.x, centroidOther.y, centroidOther.z,
							___XPointFrom(mesh->Faces + j, 0), ___YPointFrom(mesh->Faces + j, 0), ___ZPointFrom(mesh->Faces + j, 0),
							___XPointFrom(mesh->Faces + j, 1), ___YPointFrom(mesh->Faces + j, 1), ___ZPointFrom(mesh->Faces + j, 1),
							___XPointFrom(mesh->Faces + j, 2), ___YPointFrom(mesh->Faces + j, 2), ___ZPointFrom(mesh->Faces + j, 2));

						temp = centroidFirst;
						DECREASE_SEGMENT_TO_INTERVAL_3D(temp.x, temp.y, temp.z,
									centroidOther.x, centroidOther.y, centroidOther.z);
						DECREASE_SEGMENT_TO_INTERVAL_3D(centroidOther.x, centroidOther.y, centroidOther.z,
									temp.x, temp.y, temp.z);

						if (IsSegmentIntersectModel(&temp, &centroidOther, mesh) == 0)
							if (mesh->TypesOfFaces[j] == FIRST_VISION)
								mesh->TypesOfFaces[j] = DOUBLE_VISION;
							else mesh->TypesOfFaces[j] = SECOND_VISION;

					}
		}

		//else if non-triangle ...
		//...
		//..
}







__global__
void cudaToCountSecondAndDoubleFaces(CalcMesh *mesh, float *ret)
{
	//just only for triangle-polygons now
	integer i,j;
	CalcVertex centroidFirst, centroidOther, temp;
	//for {all first_vision face polygons}
	//for (i=0; i < mesh->NumberOfFaces; i++)
	i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < mesh->NumberOfFaces)
	{
		if ((mesh->TypesOfFaces[i] == FIRST_VISION)&&(mesh->Faces[i].VertexCount == 3))
		{
			GET_CENTROID(centroidFirst.x, centroidFirst.y, centroidFirst.z,
				___XPointFrom(mesh->Faces + i, 0), ___YPointFrom(mesh->Faces + i, 0), ___ZPointFrom(mesh->Faces + i, 0),
				___XPointFrom(mesh->Faces + i, 1), ___YPointFrom(mesh->Faces + i, 1), ___ZPointFrom(mesh->Faces + i, 1),
				___XPointFrom(mesh->Faces + i, 2), ___YPointFrom(mesh->Faces + i, 2), ___ZPointFrom(mesh->Faces + i, 2));

			//fo {all other_vision face polygons}
			for (j=0; j < mesh->NumberOfFaces; j++)
				if ( ( (mesh->TypesOfFaces[j] == OTHER_VISION) || (mesh->TypesOfFaces[j] == FIRST_VISION) )
					&&(mesh->Faces[j].VertexCount == 3))
					{
						GET_CENTROID(centroidOther.x, centroidOther.y, centroidOther.z,
							___XPointFrom(mesh->Faces + j, 0), ___YPointFrom(mesh->Faces + j, 0), ___ZPointFrom(mesh->Faces + j, 0),
							___XPointFrom(mesh->Faces + j, 1), ___YPointFrom(mesh->Faces + j, 1), ___ZPointFrom(mesh->Faces + j, 1),
							___XPointFrom(mesh->Faces + j, 2), ___YPointFrom(mesh->Faces + j, 2), ___ZPointFrom(mesh->Faces + j, 2));

						temp = centroidFirst;
						DECREASE_SEGMENT_TO_INTERVAL_3D(temp.x, temp.y, temp.z,
									centroidOther.x, centroidOther.y, centroidOther.z);
						DECREASE_SEGMENT_TO_INTERVAL_3D(centroidOther.x, centroidOther.y, centroidOther.z,
									temp.x, temp.y, temp.z);

						if (IsSegmentIntersectModel(&temp, &centroidOther, mesh) == 0)
							if (mesh->TypesOfFaces[j] == FIRST_VISION)
								mesh->TypesOfFaces[j] = DOUBLE_VISION;
							else mesh->TypesOfFaces[j] = SECOND_VISION;

					}
		}

		//else if non-triangle ...
		//...
		//..

	}
	*ret = 99;
}




__global__
void cudaToCountSphere(CalcMesh *mesh, float *ret)
{
        integer thread = threadIdx.x + blockIdx.x * blockDim.x;
        if (thread < mesh->NumberSphereDetalisation*(mesh->NumberSphereDetalisation-1))
        {
		integer	  i = thread / mesh->NumberSphereDetalisation + 1,	//for i=1 to i++<N
			  j = thread % mesh->NumberSphereDetalisation,		//for j=0 to j++<N
			  i_ = i-1,
			  j_ = j-1;
		if (j_==-1) j_+=mesh->NumberSphereDetalisation;

		real	  r = 3000.0,
		 	 pi = 3.14159265358979323846,
			 iphi = -pi/2.0 +  i*pi/mesh->NumberSphereDetalisation,
                        i_phi = -pi/2.0 + i_*pi/mesh->NumberSphereDetalisation,
		       jalpha = -pi +  j*2.0*pi/mesh->NumberSphereDetalisation,
		      j_alpha = -pi + j_*2.0*pi/mesh->NumberSphereDetalisation,
			sinjalpha, cosjalpha, siniphi, cosiphi,
			sinj_alpha, cosj_alpha, sini_phi, cosi_phi;
			
			sincos(iphi, &siniphi, &cosiphi);
			sincos(i_phi, &sini_phi, &cosi_phi);
			sincos(jalpha, &sinjalpha, &cosjalpha);
			sincos(j_alpha, &sinj_alpha, &cosj_alpha);
			
		CalcVertex centroid; //the sphere polygon center
		GET_CENTROID_4(	centroid.x, centroid.y, centroid.z, 
				r*cosiphi*cosjalpha, r*cosiphi*sinjalpha, r*siniphi,
				r*cosi_phi*cosjalpha, r*cosi_phi*sinjalpha, r*sini_phi,
				r*cosi_phi*cosj_alpha, r*cosi_phi*sinj_alpha, r*sini_phi,
				r*cosiphi*cosj_alpha, r*cosiphi*sinj_alpha, r*siniphi); 
		
		//change this
		//ToCountFirstFaces(&centroid, mesh);	
		
		mesh->SpherePolygonRadiosity[thread] += 0.5f;
		
		
	}
	*ret = 99;
}



//true, if no polygons between light and polygon
//__host__ __device__
//bool IsPolygonFirstFace(float *point, float *polygon, float *polygon_plane, unsigned int size,
//						  float *ArrayOfPolygon3, float *Polygon3Plane, unsigned int NumberOfPolygon3,
//						  float *ArrayOfPolygon4, float *Polygon4Plane, unsigned int NumberOfPolygon4)
//{
//	bool IntersectedModel=false;
//	int i;
//	for (int j=0; j<size; j++)						//decrease TODO
//	{
//		for (i=0; i<NumberOfPolygon3; i++)
//		{
//			if (polygon == ArrayOfPolygon3+9*i) break;
//			if(IntersectedModel=IsSegmentIntersectPolygon(point, polygon+3*j, ArrayOfPolygon3+9*i, Polygon3Plane+4*i, 3))
//				break;
//		}
//		if(!IntersectedModel)
//			for (i=0; i<NumberOfPolygon4; i++)
//			{
//				if (polygon == ArrayOfPolygon4+12*i) break;
//				if (IntersectedModel = IsSegmentIntersectPolygon(point, polygon+3*j, ArrayOfPolygon4+12*i, Polygon4Plane+4*i, 4))
//					break;
//			}
//		if (IntersectedModel) break;
//	}
//	return IntersectedModel;
//}

__global__ void GPU_tester(CalcMesh* cuda_mesh, float* ret, CalcVertex *light)
{



	//first of all we should to finish copy of our cuda CalcMesh:
	integer i;
	integer id = threadIdx.x + blockIdx.x * blockDim.x;
	//*ret = 2;
 	if (id==0)
		for (i=0; i<cuda_mesh->NumberOfFaces; i++)
		{
			cuda_mesh->Faces[i].VertexArray = cuda_mesh->VertexArray;
		}



}


void GPU_example(CalcMesh* mesh)
{
	//create cudaCalcMesh on GPU
	//i don't know how to use malloc on the gpu, i think that i don't have it on my capability 1.2

	//printf("let's go\n");

	//\F1\E4\E5\EB\E0\F2\FC \ED\EE\F0\EC\E0\EB\FC\ED\F3\FE \EE\E1\F0\E0\E1\EE\F2\EA\F3 \EE\F8\E8\E1\EE\EA

	CalcMesh *cuda_mesh, temp_mesh;

	/*CalcVertex temp_temp, *dev_temp;
	temp_temp.x = 1.1;
	temp_temp.y = 2.2;
	temp_temp.z = 3.13;


	assert( hipMalloc((void **)&dev_temp, sizeof(CalcVertex)) == hipSuccess );
	assert( hipMemcpy(dev_temp, &temp_temp, sizeof(CalcVertex), hipMemcpyHostToDevice) == hipSuccess);
	*/


	assert( hipMalloc((void **)&temp_mesh.VertexArray, mesh->NumberOfVertices*sizeof(CalcVertex)) == hipSuccess );
	assert( hipMemcpy(temp_mesh.VertexArray, mesh->VertexArray, mesh->NumberOfVertices*sizeof(CalcVertex), hipMemcpyHostToDevice) == hipSuccess);

	temp_mesh.NumberOfFaces = mesh->NumberOfFaces;
	temp_mesh.NumberOfVertices = mesh->NumberOfVertices;
	temp_mesh.NumberSphereDetalisation = mesh->NumberSphereDetalisation;


	assert( hipMalloc((void **)&temp_mesh.Faces, mesh->NumberOfFaces*sizeof(CalcFace)) == hipSuccess );
	assert( hipMemcpy(temp_mesh.Faces, mesh->Faces, mesh->NumberOfFaces*sizeof(CalcFace), hipMemcpyHostToDevice) == hipSuccess );
	//after this we need to change all temp_mesh.Faces[ i ].VertexArray to temp_mesh.VertexArray.
	//we are do it in the __global__ function

	assert( hipMalloc((void **)&temp_mesh.TypesOfFaces, mesh->NumberOfFaces*sizeof(unsigned char)) == hipSuccess );
	assert( hipMemcpy(temp_mesh.TypesOfFaces, mesh->TypesOfFaces, mesh->NumberOfFaces*sizeof(unsigned char), hipMemcpyHostToDevice) == hipSuccess );

	assert( hipMalloc((void **)&temp_mesh.Lights, sizeof(CalcVertex)) == hipSuccess );
	assert( hipMemcpy(temp_mesh.Lights, mesh->Lights, sizeof(CalcVertex), hipMemcpyHostToDevice) == hipSuccess );	
	
	assert( hipMalloc((void **)&temp_mesh.SpherePolygonRadiosity, mesh->NumberSphereDetalisation*(mesh->NumberSphereDetalisation-1)*sizeof(real)) == hipSuccess );
        assert( hipMemcpy(temp_mesh.SpherePolygonRadiosity, mesh->SpherePolygonRadiosity, mesh->NumberSphereDetalisation*(mesh->NumberSphereDetalisation-1)*sizeof(real), hipMemcpyHostToDevice) == hipSuccess );
	
	assert( hipMalloc((void **)&cuda_mesh, sizeof(CalcMesh)) == hipSuccess );
	assert( hipMemcpy(cuda_mesh, &temp_mesh, sizeof(CalcMesh), hipMemcpyHostToDevice) == hipSuccess );

	//--------------------------------------------------------

	//do something
	//temp
	/*temp_mesh->VertexArray = (CalcVertex*) calloc(mesh->NumberOfVertices, sizeof(CalcVertex));
	memcpy(temp_mesh->VertexArray, mesh->VertexArray, mesh->NumberOfVertices * sizeof(CalcVertex));
	temp_mesh->VertexArray = mesh->VertexArray;
	temp_mesh->Faces = temp_faces;
	temp_mesh->TypesOfFaces = mesh->TypesOfFaces;
*/

	//		CalcVertex light;
	//light.x = 100.0f;
	//light.y = light.z = 50.0f;


	//ToCountFirstFaces(&light, temp_mesh);

	float *temp, temp2;
	hipMalloc((void**)&temp, sizeof(float));


	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop , 0 );
	//printf("%d\n", prop.maxThreadsPerBlock);

	//CalcVertex light;
	//light.x = mesh->Lights[0].x;
	//light.y = mesh->Lights[0].y;
	//light.z = mesh->Lights[0].z;
	//DWORD time;

	GPU_tester<<<1, 1>>>(cuda_mesh, temp, mesh->Lights);

	{
		printf("DEBUG: cudaToCountFirstFaces() on %i x %i \n ",60000,prop.maxThreadsPerBlock-150);
		cudaToCountFirstFaces<<< 60000, prop.maxThreadsPerBlock-150>>>(mesh->Lights[0], cuda_mesh, temp);
		//for (integer face = 0; face < 1000; face++){
		//cudaToCountFirstFaces<<< 1, 1>>>(face, light, cuda_mesh, temp);
		printf("%s\n",hipGetErrorString(hipDeviceSynchronize()));	
		hipMemcpy(&temp2, temp, sizeof(float), hipMemcpyDeviceToHost);
		printf("DEBUG: it's resulsts %f \n", temp2);
		//}
		
		printf("DEBUG: cudaToCountSecondAndDoubleFaces() on %i x %i \n ",60000,prop.maxThreadsPerBlock-150);
		cudaToCountSecondAndDoubleFaces<<< 50000, prop.maxThreadsPerBlock-150>>>(cuda_mesh, temp);
		printf("%s\n",hipGetErrorString(hipDeviceSynchronize()));
		hipMemcpy(&temp2, temp, sizeof(float), hipMemcpyDeviceToHost);
		printf("GPU test 2:%f \n", temp2);
		
		printf("DEBUG: cudaToCountSphere() on %i x %i \n ",60000,prop.maxThreadsPerBlock-150);
                cudaToCountSphere<<< 50000, prop.maxThreadsPerBlock-150>>>(cuda_mesh, temp);
                printf("%s\n",hipGetErrorString(hipDeviceSynchronize()));
                hipMemcpy(&temp2, temp, sizeof(float), hipMemcpyDeviceToHost);
                printf("GPU test 3:%f \n", temp2);	
		
	}

	hipMemcpy(mesh->TypesOfFaces, temp_mesh.TypesOfFaces, mesh->NumberOfFaces*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(mesh->SpherePolygonRadiosity, temp_mesh.SpherePolygonRadiosity, mesh->NumberSphereDetalisation*(mesh->NumberSphereDetalisation-1)*sizeof(real), hipMemcpyDeviceToHost);
	
//DEBUG
	{
/*		CalcVertex point2;
BUG Ham		point2.x = -34.475559;
		point2.y = 143.815369;
		point2.z = -271.077057;
		if (IsSegmentIntersectPolygon(&light, &point2, &(mesh->Faces[23127]))) printf("Yes, intersection\n");
		else printf("No intersection\n");
*/	
		//ON_ONE_SIDE(point2.x, point2.y,
		//	 p2x, p2y,
		//	 p1x_line, p1y_line,
		//	 p2x_line, p2y_line);
		//printf("%f\n", mesh->Faces[23127].VertexArray[ mesh->Faces[23127].VertexIndices[2] ].z);	
	}

	int j;
	//from 2000 to 2200 - good faces for experiments in  ham
	//for (j=800; j<1000; j++)		
	//mesh->TypesOfFaces[2150] = 2;
	//mesh->TypesOfFaces[23127] = 2;

/*CPU	{
	//time = GetTickCount();
		ToCountFirstFaces(&light, mesh);
		ToCountSecondAndDoubleFaces(mesh);		
	//time = time - GetTickCount();
//	printf("time: %i\n", time);
	}
*/


	//--------------------------------------------------------
	//free memory
	hipFree(cuda_mesh);
	hipFree(temp_mesh.TypesOfFaces);
	hipFree(temp_mesh.Faces);
	hipFree(temp_mesh.VertexArray);


}
